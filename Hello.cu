#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort) exit(code);
  }
}

__global__ void k() {
  printf("hello %u!\n", threadIdx.x);
}

int main(void) {
  k<<<3, 32>>>();
  hipDeviceSynchronize();
  gpuErrchk(hipPeekAtLastError());
  return 0;
}
